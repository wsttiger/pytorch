
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricFullConvolution.cu"
#else

void THNN_(VolumetricFullConvolution_updateOutput)(
       THCState *state,
       THCTensor  *input,
       THCTensor  *output,
       THCTensor  *weight,
       THCTensor  *bias,
       THCTensor  *finput,
       THCTensor  *fgradInput,
       int dT, int dW, int dH,
       int padT, int padW, int padH,
       int adjT, int adjW, int adjH)
{
#if !defined(__HIP_PLATFORM_HCC__)
  THNN_(VolumetricFullDilatedConvolution_updateOutput)(
       state, input, output, weight, bias, finput, fgradInput,
       dT, dW, dH, padT, padW, padH, 1, 1, 1, adjT, adjW, adjH);
#endif
}

void THNN_(VolumetricFullConvolution_updateGradInput)(
       THCState *state,
       THCTensor  *input,
       THCTensor  *gradOutput,
       THCTensor  *gradInput,
       THCTensor  *weight,
       THCTensor  *finput,
       THCTensor  *fgradInput,
       int dT, int dW, int dH,
       int padT, int padW, int padH,
       int adjT, int adjW, int adjH)
{
#if !defined(__HIP_PLATFORM_HCC__)
  THNN_(VolumetricFullDilatedConvolution_updateGradInput)(
       state, input, gradOutput, gradInput, weight, finput, fgradInput,
       dT, dW, dH, padT, padW, padH, 1, 1, 1, adjT, adjW, adjH);
#endif
}


void THNN_(VolumetricFullConvolution_accGradParameters)(
           THCState *state,
           THCTensor  *input,
           THCTensor  *gradOutput,
           THCTensor  *gradWeight,
           THCTensor  *gradBias,
           THCTensor  *finput,
           THCTensor  *fgradInput,
           int dT, int dW, int dH,
           int padT, int padW, int padH,
           int adjT, int adjW, int adjH,
           accreal scale_)
{
#if !defined(__HIP_PLATFORM_HCC__)
  THNN_(VolumetricFullDilatedConvolution_accGradParameters)(
       state, input, gradOutput, gradWeight, gradBias, finput, fgradInput,
       dT, dW, dH, padT, padW, padH, 1, 1, 1, adjT, adjW, adjH, scale_);
#endif
}

#endif
