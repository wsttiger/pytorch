
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorIndex.cu"
#else

// Check tensor dimensions for index operations, and return the slice size.
// src can be nullptr in case of indexFill: in that case it is ignored.
static ptrdiff_t THCTensor_(getSliceSize)(THCState *state, THCTensor *dst,
                                          int dim,
                                          THCudaLongTensor *index,
                                          THCTensor *src)
{
  int dstDims = THCTensor_(nDimension)(state, dst);
  int srcDims = (src == nullptr) ? dstDims : THCTensor_(nDimension)(state, src);

  THArgCheck(THCudaLongTensor_nDimension(state, index) == 1, 4,
             "expecting vector of indices");
  THArgCheck(dim >= 0 && dim < dstDims, 2, "Indexing dim is out of bounds");

  ptrdiff_t dstSliceSize = 1;
  for (int d = 0; d < dstDims; d++) {
    if (d != dim) {
      dstSliceSize *= dst->size[d];
    }
  }

  if (src == nullptr) return dstSliceSize;

  THArgCheck(dim < srcDims, 3, "Indexing dim is out of bounds");
  THArgCheck(THCudaLongTensor_nElement(state, index) == src->size[dim], 4,
             "length of src.size[dim] is not equal to length of indices");

  ptrdiff_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (int d = 0; d < srcDims; d++) {
    if (d != dim) {
      srcSliceSize *= src->size[d];
      if (!mismatch && dst->size[d] != src->size[d]) mismatch = true;
    }
  }

  THArgCheck(dstSliceSize == srcSliceSize, 2,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    static bool warningShown = false;
    if (!warningShown) {
      warningShown = true;
      fprintf(stderr,
              "Warning: source/destination slices have same size but different "
              "shape for an index operation.  This behavior is deprecated.\n");
    }
  }

  return dstSliceSize;
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
bool THCTensor_(indexShouldBeMajor)(TensorInfo<real, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (int i = 0; i < info.dims; ++i) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

void THCTensor_(indexCopy_long)(THCState *state, THCTensor *dst, int dim, THLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexCopy)(state, dst, dim, indices_, src);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexCopy)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize = THCTensor_(getSliceSize)(state, dst, dim, indices, src);
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstCopyDimSize = THCTensor_(size)(state, dst, dim);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);
  cudaStream_t stream = THCState_getCurrentStream(state);
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#if defined(__HIP_PLATFORM_HCC__)
#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)      \
  hipLaunchKernelGGL(                                                  \
  (indexCopySmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>), \
      smallIndexGrid, smallIndexBlock, 0, stream,                      \
      dstInfo, srcInfo, indicesInfo,                                   \
      dstCopyDim, srcCopyDim, sliceSize, dstCopyDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                           \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)     \
  hipLaunchKernelGGL(                                            \
  (indexCopyLargeIndex<TENSOR_TYPE, TYPE,                        \
                      DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>), \
      largeIndexGrid, largeIndexBlock, 0, stream,                \
      dstInfo, srcInfo, indicesInfo,                             \
      dstCopyDim, srcCopyDim, srcTotalSize,                      \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                   \
      dstCopyDimSize);
#else
#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)   \
  indexCopySmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(               \
      dstInfo, srcInfo, indicesInfo,                                \
      dstCopyDim, srcCopyDim, sliceSize, dstCopyDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                         \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)   \
  indexCopyLargeIndex<TENSOR_TYPE, TYPE,                       \
                      DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(          \
      dstInfo, srcInfo, indicesInfo,                           \
      dstCopyDim, srcCopyDim, srcTotalSize,                    \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                 \
      dstCopyDimSize);
#endif

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, src) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<THCTensor, unsigned int>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstCopyDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<THCTensor, uint64_t>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(take)(THCState *state, THCTensor *dst, THCTensor *src, THCudaLongTensor *index)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  THArgCheck(THCTensor_(nDimension)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimension)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(!(THCTensor_(nDimension)(state, src) == 0 && THCudaLongTensor_nDimension(state, index) != 0), 2,
             "tried to take from an empty tensor");

  THCTensor_(resizeNd)(state, dst, index->nDimension, index->size, NULL);

  // dispatchTakePut only handles non-empty tensors;
  if (index->nDimension > 0) {
    dispatchTakePut<real, TensorTakeOp>(state, src, dst, index);
  }
}

static void THCTensor_(sort_indices)(THCState *state, THCudaLongTensor *index, THCTensor *src) {
  THCThrustAllocator thrustAlloc(state);

  auto index_iter = thrust::device_ptr<int64_t>(THCudaLongTensor_data(state, index));
  auto src_iter = thrust::device_ptr<real>(THCTensor_(data)(state, src));
  auto numel = THCTensor_(numel)(state, src);

  thrust::sort_by_key(
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
    index_iter, index_iter + numel,
    src_iter, ThrustLTOp<int64_t>());
}

void THCTensor_(put)(THCState *state, THCTensor *dst, THCudaLongTensor *index, THCTensor *src, int accumulate)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  ptrdiff_t dstSize = THCTensor_(nElement)(state, dst);
  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, index);
  THArgCheck(THCTensor_(nElement)(state, src) == numIndices,
    3, "src should have the same number of elements as index");

  THArgCheck(THCTensor_(nDimension)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimension)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  if (numIndices == 0) {
    return;
  }

  if (accumulate) {
    // wrap indices so to replace negative indices
    THCudaLongTensor* sorted_index = THCudaLongTensor_new(state);
    THCudaLongTensor_resizeAs(state, sorted_index, index);
    THC_pointwiseApply2(state, sorted_index, index, WrapIndexOp(dstSize));

    THCTensor* sorted_src = THCTensor_(newClone)(state, src);

    THCTensor_(sort_indices)(state, sorted_index, sorted_src);
    dispatchTakePut<real, TensorPutAccumulateOp>(state, dst, sorted_src, sorted_index);

    THCTensor_(free)(state, sorted_src);
    THCudaLongTensor_free(state, sorted_index);
  } else {
    dispatchTakePut<real, TensorPutOp>(state, dst, src, index);
  }
}

void THCTensor_(indexAdd_long)(THCState *state, THCTensor *dst, int dim, THLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexAdd)(state, dst, dim, indices_, src);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexAdd)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize = THCTensor_(getSliceSize)(state, dst, dim, indices, src);
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstAddDimSize = THCTensor_(size)(state, dst, dim);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);
  cudaStream_t stream = THCState_getCurrentStream(state);
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#if defined(__HIP_PLATFORM_HCC__)
#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)  \
  hipLaunchKernelGGL(                                              \
    (indexAddSmallIndex<TENSOR_TYPE, TYPE,                         \
                        DST_DIM, SRC_DIM, IDX_DIM>),               \
      smallIndexGrid, smallIndexBlock, 0, stream,                  \
      dstInfo, srcInfo, indicesInfo,                               \
      dstAddDim, srcAddDim, sliceSize, dstAddDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                             \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)       \
  hipLaunchKernelGGL(                                              \
  (indexAddLargeIndex<TENSOR_TYPE, TYPE,                           \
                     DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>),    \
      largeIndexGrid, largeIndexBlock, 0, stream,                  \
      dstInfo, srcInfo, indicesInfo,                               \
      dstAddDim, srcAddDim, srcTotalSize,                          \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                     \
      dstAddDimSize);
#else
#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)  \
  indexAddSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(              \
      dstInfo, srcInfo, indicesInfo,                               \
      dstAddDim, srcAddDim, sliceSize, dstAddDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                             \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)       \
  indexAddLargeIndex<TENSOR_TYPE, TYPE,                            \
                     DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>      \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(              \
      dstInfo, srcInfo, indicesInfo,                               \
      dstAddDim, srcAddDim, srcTotalSize,                          \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                     \
      dstAddDimSize);
#endif

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, src) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstAddDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstAddDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<THCTensor, unsigned int>(state, src);
    int srcAddDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcAddDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstAddDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstAddDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstAddDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<THCTensor, uint64_t>(state, src);
    int srcAddDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcAddDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

void THCTensor_(indexFill_long)(THCState *state, THCTensor *dst, int dim, THLongTensor *indices, real val)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, dst));

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexFill)(state, dst, dim, indices_, val);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexFill)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, real val)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, dst));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));
  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize =
    THCTensor_(getSliceSize)(state, dst, dim, indices, nullptr);
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t dstFillDimSize = THCTensor_(size)(state, dst, dim);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);
  cudaStream_t stream = THCState_getCurrentStream(state);
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#if defined(__HIP_PLATFORM_HCC__)
#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM)            \
  hipLaunchKernelGGL(                                               \
  (indexFillSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM>),       \
      smallIndexGrid, smallIndexBlock, 0, stream,                   \
      dstInfo, indicesInfo,                                         \
      dstFillDim, sliceSize, dstFillDimSize, val);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR)      \
  hipLaunchKernelGGL(                                                       \
  (indexFillLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR>), \
      largeIndexGrid, largeIndexBlock, 0, stream,                           \
      dstInfo, indicesInfo,                                                 \
      dstFillDim, sliceSize * numIndices,                                   \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                              \
      dstFillDimSize, val);
#else
#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM)          \
  indexFillSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM>        \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(             \
      dstInfo, indicesInfo,                                       \
      dstFillDim, sliceSize, dstFillDimSize, val);

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR)   \
  indexFillLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                    \
      dstInfo, indicesInfo,                                              \
      dstFillDim, sliceSize * numIndices,                                \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                           \
      dstFillDimSize, val);
#endif

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, -2);
      } else if (dstInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, -2);
      } else if (dstInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstFillDim);

      if (dstInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, -2, true);
      } else if (dstInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}


void THCTensor_(indexSelect_long)(THCState *state, THCTensor *dst, THCTensor *src, int dim, THLongTensor *indices)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THArgCheck(indices->nDimension <= 1, 3, "Index is supposed to be an empty tensor or a vector");

  THCudaLongTensor *indices_ = THCudaLongTensor_newWithSize1d(state, indices->size[0]);
  THCudaLongTensor_copyLong(state, indices_, indices);

  THCTensor_(indexSelect)(state, dst, src, dim, indices_);

  THCudaLongTensor_free(state, indices_);
}

void THCTensor_(indexSelect)(THCState *state, THCTensor *dst, THCTensor *src, int dim, THCudaLongTensor *indices)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, dst, src, indices));

  int dims = THCTensor_(nDimension)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimension)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);

  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  int srcDims = THCTensor_(nDimension)(state, src);
  cudaStream_t stream = THCState_getCurrentStream(state);

  THArgCheck(THCudaLongTensor_nDimension(state, indices) <= 1, 3,
             "Index is supposed to be an empty tensor or a vector");
  THArgCheck(dim < srcDims, 4, "Indexing dim is out of bounds");
  THArgCheck(srcDims > 0, 2, "Source tensor is empty");

  THLongStorage *newSize;

  if (numIndices == 0) {
    newSize = THCTensor_(newSizeOf)(state, src);
    THLongStorage_set(newSize, 0, numIndices);
    THCTensor_(resize)(state, dst, newSize, NULL);
    THLongStorage_free(newSize);
    return;
  }

  newSize = THCTensor_(newSizeOf)(state, src);
  THLongStorage_set(newSize, dim, numIndices);
  THCTensor_(resize)(state, dst, newSize, NULL);
  THLongStorage_free(newSize);

  int indContig = THCudaLongTensor_isContiguous(state, indices);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t srcSelectDimSize = THCTensor_(size)(state, src, dim);
  ptrdiff_t sliceSize = dstTotalSize / numIndices;

  int mpc = THCState_getCurrentDeviceProperties(state)->multiProcessorCount;

#if defined(__HIP_PLATFORM_HCC__)
  #define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)\
    hipLaunchKernelGGL(                                            \
    (indexSelectSmallIndex<TENSOR_TYPE, TYPE, DST_DIM,             \
                           SRC_DIM, IDX_DIM>),                     \
        smallIndexGrid, smallIndexBlock, 0, stream,                \
        dstInfo, srcInfo, indicesInfo,                             \
        dstSelectDim, srcSelectDim, sliceSize, srcSelectDimSize);

  #define LARGE_INDEX(TENSOR_TYPE, TYPE,                           \
                      DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)     \
    hipLaunchKernelGGL(                                            \
    (indexSelectLargeIndex<TENSOR_TYPE, TYPE, DST_DIM,             \
                           SRC_DIM, IDX_DIM, IDX_IS_MAJOR>),       \
       largeIndexGrid, largeIndexBlock, 0, stream,                 \
        dstInfo, srcInfo, indicesInfo,                             \
        dstSelectDim, srcSelectDim, dstTotalSize,                  \
        (IDX_IS_MAJOR) ? sliceSize : numIndices,                   \
        srcSelectDimSize);
  #else
  #define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)     \
    indexSelectSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM> \
      <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                 \
        dstInfo, srcInfo, indicesInfo,                                  \
        dstSelectDim, srcSelectDim, sliceSize, srcSelectDimSize);

  #define LARGE_INDEX(TENSOR_TYPE, TYPE,                                 \
                      DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)           \
    indexSelectLargeIndex<TENSOR_TYPE, TYPE,                             \
                          DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>       \
      <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                  \
        dstInfo, srcInfo, indicesInfo,                                   \
        dstSelectDim, srcSelectDim, dstTotalSize,                        \
        (IDX_IS_MAJOR) ? sliceSize : numIndices,                         \
        srcSelectDimSize);
#endif

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, dst) &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, src) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    TensorInfo<real, unsigned int> dstInfo =
      getTensorInfo<THCTensor, unsigned int>(state, dst);
    int dstSelectDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstSelectDim);

    TensorInfo<real, unsigned int> srcInfo =
      getTensorInfo<THCTensor, unsigned int>(state, src);
    int srcSelectDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcSelectDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(real, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(real, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(real, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(real, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstSelectDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(real, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(real, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(real, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<real, uint64_t> dstInfo =
      getTensorInfo<THCTensor, uint64_t>(state, dst);
    int dstSelectDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstSelectDim);

    TensorInfo<real, uint64_t> srcInfo =
      getTensorInfo<THCTensor, uint64_t>(state, src);
    int srcSelectDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcSelectDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(real, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

#define MAX_ADVINDEX_CALC_DIMS 5

void THCTensor_(calculateAdvancedIndexingOffsets)(
  THCState *state,
  THCudaLongTensor *output,
  THCTensor *indexed,
  ptrdiff_t baseOffset,
  THCudaLongTensor **indexers)
{
  int ndim = THCTensor_(nDimension)(state, indexed);
  THAssert(ndim <= MAX_ADVINDEX_CALC_DIMS);

  // Assert all Tensors are on the same GPU, and that the indexing Tensors are
  // contiguous
  for (int i = 0; i < ndim; ++i) {
    if (indexers[i] != NULL) {
      THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, output, indexers[i]));
      THAssert(THCudaLongTensor_isContiguous(state, indexers[i]));
    }
  }

  // Set grid, block dims
  ptrdiff_t nElement = THCudaLongTensor_nElement(state, output);
  const dim3 block = getApplyBlock();
  dim3 grid;
  THAssert(getApplyGrid(state, nElement, grid));

#if defined(__HIP_PLATFORM_HCC__)
  #define HANDLE_CASE(INDEX_TYPE, DIMS)                                                           \
  {                                                                                               \
    LinearIndexCalcData<INDEX_TYPE, DIMS> data;                                                   \
    for (int i = 0; i < DIMS; ++i) {                                                              \
      data.baseSizes[i] = THCTensor_(size)(state, indexed, i);                                    \
      data.sizes[i] = indexers[i] != NULL ?                                                       \
        THCudaLongTensor_nElement(state, indexers[i]) :                                           \
          THCTensor_(size)(state, indexed, i);                                                    \
      data.strides[i] = THCTensor_(stride)(state, indexed, i);                                    \
      data.advIndexTensors[i] = indexers[i] != NULL ?                                             \
        THCudaLongTensor_data(state, indexers[i]) : NULL;                                         \
    }                                                                                             \
                                                                                                  \
    hipLaunchKernelGGL(                                                                           \
      (calculateLinearIndices<INDEX_TYPE, DIMS>),                                                 \
        grid, block, 0, THCState_getCurrentStream(state),                                         \
        THCudaLongTensor_data(state, output),                                                     \
        nElement,                                                                                 \
        baseOffset,                                                                               \
        data                                                                                      \
    );                                                                                            \
  }
#else
  #define HANDLE_CASE(INDEX_TYPE, DIMS)                                                           \
  {                                                                                               \
    LinearIndexCalcData<INDEX_TYPE, DIMS> data;                                                   \
    for (int i = 0; i < DIMS; ++i) {                                                              \
      data.baseSizes[i] = THCTensor_(size)(state, indexed, i);                                    \
      data.sizes[i] = indexers[i] != NULL ?                                                       \
        THCudaLongTensor_nElement(state, indexers[i]) :                                           \
          THCTensor_(size)(state, indexed, i);                                                    \
      data.strides[i] = THCTensor_(stride)(state, indexed, i);                                    \
      data.advIndexTensors[i] = indexers[i] != NULL ?                                             \
        THCudaLongTensor_data(state, indexers[i]) : NULL;                                         \
    }                                                                                             \
                                                                                                  \
    calculateLinearIndices<INDEX_TYPE, DIMS>                                                      \
      <<<grid, block, 0, THCState_getCurrentStream(state)>>>(                                     \
      THCudaLongTensor_data(state, output),                                                       \
      nElement,                                                                                   \
      baseOffset,                                                                                 \
      data                                                                                        \
    );                                                                                            \
  }
#endif

#define RUN_T(INDEX_TYPE)         \
  switch (ndim) {                 \
    case 1:                       \
      HANDLE_CASE(INDEX_TYPE, 1)  \
      break;                      \
    case 2:                       \
      HANDLE_CASE(INDEX_TYPE, 2)  \
      break;                      \
    case 3:                       \
      HANDLE_CASE(INDEX_TYPE, 3)  \
      break;                      \
    case 4:                       \
      HANDLE_CASE(INDEX_TYPE, 4)  \
      break;                      \
    case 5:                       \
      HANDLE_CASE(INDEX_TYPE, 5)  \
      break;                      \
    default:                      \
      THAssert(false);            \
}

  if (TensorUtils<THCTensor>::canUse32BitIndexMath(state, indexed)) {
    RUN_T(unsigned int);
  } else {
    RUN_T(uint64_t);
  }

#undef HANDLE_CASE
#undef RUN_T

  THCudaCheck(cudaGetLastError());
}

#undef MAX_ADVINDEX_CALC_DIMS

#endif
